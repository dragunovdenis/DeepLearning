#include "hip/hip_runtime.h"
//Copyright (c) 2022 Denys Dragunov, dragunovdenis@gmail.com
//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files(the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
//copies of the Software, and to permit persons to whom the Software is furnished
//to do so, subject to the following conditions :

//The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
//INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
//PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
//HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
//OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
//SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include "CudaVector.cuh"
#include <hip/hip_runtime.h>
#include "CudaUtils.cuh"
#include <thrust/execution_policy.h>
#include <thrust/equal.h>

namespace DeepLearning
{
	/// <summary>
	/// Frees the allocated memory
	/// </summary>
	void CudaVector::free()
	{
		if (_data != nullptr)
		{
			gpuErrchk(hipFree(_data));
			_data = nullptr;
		}

		_dim = 0;
	}

	void CudaVector::resize(const std::size_t& new_size)
	{
		if (size() != new_size)
		{
			free();
			_dim = new_size;
			_data = CudaUtils::cuda_allocate<Real>(_dim);
		}
	}

	void CudaVector::assign(const BasicCudaCollection& source)
	{
		resize(source.size());
		CudaUtils::cuda_copy_device2device(begin(), source.begin(), _dim);
	}

	void CudaVector::assign(const BasicCollection& source)
	{
		resize(source.size());
		CudaUtils::cuda_copy_host2device(begin(), source.begin(), _dim);
	}

	std::size_t CudaVector::size() const
	{
		return _dim;
	}

	Index3d CudaVector::size_3d() const
	{
		return { 1ull, 1ull, _dim };
	}

	Vector CudaVector::to_host() const
	{
		Vector result(size(), false/*assign zero*/);
		CudaUtils::cuda_copy_device2host(result.begin(), begin(), size());

		return result;
	}

	void CudaVector::msgpack_unpack(msgpack::object const& msgpack_o)
	{
		Vector proxy;
		msgpack::type::make_define_array(proxy).msgpack_unpack(msgpack_o);
		assign(proxy);
	}

	CudaVector::CudaVector(const std::size_t dim, const bool assign_zero) : _dim(dim)
	{
		_data = CudaUtils::cuda_allocate<Real>(_dim);

		if (assign_zero)
			CudaUtils::fill_zero(_data, _dim);
	}

	CudaVector::CudaVector(const Index3d& size, const bool assign_zero) : 
		CudaVector(size.z, assign_zero)
	{
		if (size.x != 1ll || size.y != 1ll)
			throw std::exception("Invalid input size");
	}

	CudaVector::CudaVector(const CudaVector& vec) :
		CudaVector(vec.size(), false /*fill zero*/)
	{
		assign(vec);
	}

	CudaVector::CudaVector(const std::size_t dim, const Real range_begin, const Real range_end) :
		CudaVector(dim, false /*assign zero*/)
	{
		uniform_random_fill(range_begin, range_end);
	}

	CudaVector::CudaVector(CudaVector&& vec) noexcept : _dim(vec._dim)
	{
		_data = vec._data;
		vec.abandon_resources();
	}

	CudaVector& CudaVector::operator=(const CudaVector& vec)
	{
		assign(vec);
		return *this;
	}

	CudaVector::~CudaVector()
	{
		free();
	}

	std::size_t CudaVector::dim() const
	{
		return _dim;
	}

	void CudaVector::abandon_resources()
	{
		_data = nullptr;
		free();
	}

	CudaVector& CudaVector::operator += (const CudaVector& vec)
	{
		add(vec);
		return *this;
	}

	CudaVector& CudaVector::operator -= (const CudaVector& vec)
	{
		sub(vec);
		return *this;
	}

	CudaVector& CudaVector::operator *= (const Real& scalar)
	{
		mul(scalar);
		return *this;
	}

	bool CudaVector::operator == (const CudaVector & vect) const
	{
		return size() == vect.size() &&
			   thrust::equal(thrust::device, begin(), end(), vect.begin());
	}

	bool CudaVector::operator !=(const CudaVector& vect) const
	{
		return !(*this == vect);
	}

	CudaVector CudaVector::random(const std::size_t dim, const Real range_begin, const Real range_end)
	{
		return CudaVector(dim, range_begin, range_end);
	}

	void CudaVector::log(const std::filesystem::path& file_name) const
	{
		to_host().log(file_name);
	}

	CudaVector operator + (const CudaVector& vec1, const CudaVector& vec2)
	{
		auto result = vec1;
		return result += vec2;
	}

	CudaVector operator -(const CudaVector& vec1, const CudaVector& vec2)
	{
		auto result = vec1;
		return result -= vec2;
	}

	CudaVector operator *(const CudaVector& vec, const Real& scalar)
	{
		auto result = vec;
		return result *= scalar;
	}

	CudaVector operator *(const Real& scalar, const CudaVector& vec)
	{
		return vec * scalar;
	}
}