#include "hip/hip_runtime.h"
//Copyright (c) 2022 Denys Dragunov, dragunovdenis@gmail.com
//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files(the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
//copies of the Software, and to permit persons to whom the Software is furnished
//to do so, subject to the following conditions :

//The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
//INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
//PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
//HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
//OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
//SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include "CudaVector.cuh"
#include <hip/hip_runtime.h>
#include "CudaUtils.cuh"
#include <thrust/execution_policy.h>
#include <thrust/equal.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include "CudaArray.cuh"
#include <thrust/scatter.h>
#include <thrust/iterator/constant_iterator.h>

namespace DeepLearning
{
	/// <summary>
	/// Frees the allocated memory
	/// </summary>
	void CudaVector::free()
	{
		if (_data != nullptr)
		{
			CudaUtils::cuda_free(_data);
			_data = nullptr;
		}

		_dim = 0;
		_capacity = 0;
	}

	void CudaVector::resize(const std::size_t& new_size)
	{
		if (_capacity < new_size)
		{
			free();
			_data = CudaUtils::cuda_allocate<Real>(new_size);
			_capacity = new_size;
		}

		_dim = new_size;
	}

	void CudaVector::resize(const Index3d& size_3d)
	{
		if (size_3d.x != 1ll || size_3d.y != 1ll)
			throw std::exception("Invalid input data");

		resize(size_3d.z);
	}


	void CudaVector::assign(const BasicCudaCollection& source)
	{
		resize(source.size());
		CudaUtils::cuda_copy_device2device(begin(), source.begin(), _dim);
	}

	void CudaVector::assign(const BasicCollection& source)
	{
		resize(source.size());
		CudaUtils::cuda_copy_host2device(begin(), source.begin(), _dim);
	}

	std::size_t CudaVector::size() const
	{
		return _dim;
	}

	std::size_t CudaVector::capacity() const
	{
		return _capacity;
	}

	Index3d CudaVector::size_3d() const
	{
		return { 1ull, 1ull, _dim };
	}

	Vector CudaVector::to_host() const
	{
		Vector result(size(), false/*assign zero*/);
		CudaUtils::cuda_copy_device2host(result.begin(), begin(), size());

		return result;
	}

	void CudaVector::msgpack_unpack(msgpack::object const& msgpack_o)
	{
		Vector proxy;
		msgpack::type::make_define_array(proxy).msgpack_unpack(msgpack_o);
		assign(proxy);
	}

	CudaVector::CudaVector(const std::size_t dim, const bool assign_zero)
	{
		resize(dim);

		if (assign_zero)
			CudaUtils::fill_zero(_data, _dim);
	}

	CudaVector::CudaVector(const Index3d& size, const bool assign_zero) : 
		CudaVector(size.z, assign_zero)
	{
		if (size.x != 1ll || size.y != 1ll)
			throw std::exception("Invalid input size");
	}

	CudaVector::CudaVector(const CudaVector& vec) :
		CudaVector(vec.size(), false /*fill zero*/)
	{
		assign(vec);
	}

	CudaVector::CudaVector(const std::size_t dim, const Real range_begin, const Real range_end) :
		CudaVector(dim, false /*assign zero*/)
	{
		uniform_random_fill(range_begin, range_end);
	}

	CudaVector::CudaVector(CudaVector&& vec) noexcept : _dim(vec._dim), _capacity(vec._capacity)
	{
		_data = vec._data;
		vec.abandon_resources();
	}

	CudaVector& CudaVector::operator=(const CudaVector& vec)
	{
		if (this != &vec)
			assign(vec);

		return *this;
	}

	CudaVector& CudaVector::operator=(CudaVector&& vec) noexcept
	{
		if (this != &vec)
		{
			free();
			_dim = vec._dim;
			_capacity = vec._capacity;
			_data = vec._data;
			vec.abandon_resources();
		}

		return *this;
	}

	CudaVector::~CudaVector()
	{
		free();
	}

	std::size_t CudaVector::dim() const
	{
		return _dim;
	}

	void CudaVector::abandon_resources()
	{
		_data = nullptr;
		free();
	}

	CudaVector& CudaVector::operator += (const CudaVector& vec)
	{
		add(vec);
		return *this;
	}

	CudaVector& CudaVector::operator -= (const CudaVector& vec)
	{
		sub(vec);
		return *this;
	}

	CudaVector& CudaVector::operator *= (const Real& scalar)
	{
		mul(scalar);
		return *this;
	}

	bool CudaVector::operator == (const CudaVector & vect) const
	{
		return size() == vect.size() &&
			   thrust::equal(thrust::cuda::par.on(hipStreamPerThread), begin(), end(), vect.begin());
	}

	bool CudaVector::operator !=(const CudaVector& vect) const
	{
		return !(*this == vect);
	}

	CudaVector CudaVector::random(const std::size_t dim, const Real range_begin, const Real range_end)
	{
		return CudaVector(dim, range_begin, range_end);
	}

	void CudaVector::log(const std::filesystem::path& file_name) const
	{
		to_host().log(file_name);
	}

	CudaVector operator + (const CudaVector& vec1, const CudaVector& vec2)
	{
		auto result = vec1;
		return result += vec2;
	}

	CudaVector operator -(const CudaVector& vec1, const CudaVector& vec2)
	{
		auto result = vec1;
		return result -= vec2;
	}

	CudaVector operator *(const CudaVector& vec, const Real& scalar)
	{
		auto result = vec;
		return result *= scalar;
	}

	CudaVector operator *(const Real& scalar, const CudaVector& vec)
	{
		return vec * scalar;
	}

	void CudaVector::generate_with_random_selection_map(const std::size_t& selected_cnt, CudaArray<int>& aux_collection)
	{
		if (selected_cnt >= size())
		{
			fill(Real(1));
			return;
		}

		aux_collection.resize(size());
		thrust::sequence(thrust::cuda::par.on(hipStreamPerThread), aux_collection.begin(), aux_collection.end(), 0);
		uniform_random_fill(Real(-1), Real(1)); //fill the current collection with random values
		//and use it as a key collection in the following sorting procedure
		thrust::sort_by_key(thrust::cuda::par.on(hipStreamPerThread), begin(), end(), aux_collection.begin());

		const auto one_iterator = thrust::make_constant_iterator(Real(1));
		thrust::scatter(thrust::cuda::par.on(hipStreamPerThread), one_iterator, one_iterator + static_cast<int>(selected_cnt),
			aux_collection.begin(), begin());

		const auto zero_iterator = thrust::make_constant_iterator(Real(0));
		thrust::scatter(thrust::cuda::par.on(hipStreamPerThread), zero_iterator, zero_iterator + static_cast<int>(size() - selected_cnt),
			aux_collection.begin() + static_cast<int>(selected_cnt), begin());
	}

	void CudaVector::generate_with_random_selection_map(const std::size_t& selected_cnt)
	{
		CudaArray<int> aux_collection;
		generate_with_random_selection_map(selected_cnt, aux_collection);
	}

}