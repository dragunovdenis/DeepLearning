//Copyright (c) 2022 Denys Dragunov, dragunovdenis@gmail.com
//Permission is hereby granted, free of charge, to any person obtaining a copy
//of this software and associated documentation files(the "Software"), to deal
//in the Software without restriction, including without limitation the rights
//to use, copy, modify, merge, publish, distribute, sublicense, and /or sell
//copies of the Software, and to permit persons to whom the Software is furnished
//to do so, subject to the following conditions :

//The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

//THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
//INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
//PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
//HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
//OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
//SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

#include "CostFunctionHelperCuda.cuh"
#include "BasicCudaCollection.cuh"
#include "thrust/reduce.h"
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <nvfunctional>
#include "CudaVector.cuh"
#include <Math/CostFunctionFactory.h>

namespace DeepLearning
{
	Real CostFunctionHelperCuda::evaluate_cost(const BasicCudaCollection& output, const BasicCudaCollection& reference, const CostFunctionId id)
	{
		thread_local CudaVector temp;
		temp.resize(output.size());

		thrust::transform(thrust::cuda::par.on(hipStreamPerThread), output.begin(), output.end(), reference.begin(), temp.begin(),
			[id] __device__(const auto&x, const auto& y) {
			const auto func = CostFunctionFactory::make<nvstd::function<Real(Real, Real)>>(id);
			return func(x, y);
		});

		return thrust::reduce(thrust::cuda::par.on(hipStreamPerThread), temp.begin(), temp.end(), static_cast<Real>(0), thrust::plus<Real>());
	}

	Real CostFunctionHelperCuda::evaluate_cost_and_gradient(BasicCudaCollection& output, const BasicCudaCollection& reference, const CostFunctionId id)
	{
		//TODO: think about more efficient solution (although this method is not supposed to be used in the training of a neural network)
		const auto func_val = evaluate_cost(output, reference, id);
		evaluate_gradient(output, reference, id);

		return func_val;
	}

	void CostFunctionHelperCuda::evaluate_gradient(BasicCudaCollection& output, const BasicCudaCollection& reference, const CostFunctionId id)
	{
		thrust::transform(thrust::cuda::par.on(hipStreamPerThread), output.begin(), output.end(), reference.begin(), output.begin(),
			[id] __device__(const auto & x, const auto & ref) {
			const auto func = CostFunctionFactory::make<nvstd::function<dual<Real>(dual<Real>, Real)>>(id);
			return  func({ x, static_cast<Real>(1) }, ref).Dual()[0];
		});
	}
}